#include "hip/hip_runtime.h"
#include "matmul.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <climits>


#define CUDA_OK(x) do {                             \
    hipError_t err__ = (x);                        \
    if (err__ != hipSuccess) {                     \
        std::fprintf(stderr, "CUDA: %s at %s:%d\n", \
                     hipGetErrorString(err__),     \
                     __FILE__, __LINE__);           \
        return 1;                                   \
    }                                               \
} while (0)

__global__ void matmul_kernel(const double* __restrict__ A,
                              const double* __restrict__ B,
                              double* __restrict__ C,
                              int M, int K, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        double sum = 0.0;
        int a_base = row * K;
        for (int p = 0; p < K; ++p) {
            sum += A[a_base + p] * B[p * N + col];
        }
        C[row * N + col] = sum;
    }
}

// Exposed C API (no name mangling)
extern "C"
int matmul_cuda(const double* A_h, const double* B_h, double* C_h,
                size_t m, size_t k, size_t n)
{

    // Guard against size_t > INT_MAX for kernel params
    if (m > INT_MAX || k > INT_MAX || n > INT_MAX) {
        std::fprintf(stderr, "Dims too large for this simple kernel.\n");
        return 2;
    }
    int M = static_cast<int>(m);
    int K = static_cast<int>(k);
    int N = static_cast<int>(n);

    size_t bytesA = m * k * sizeof(double);
    size_t bytesB = k * n * sizeof(double);
    size_t bytesC = m * n * sizeof(double);

    double *A_d = nullptr, *B_d = nullptr, *C_d = nullptr;
    CUDA_OK(hipMalloc(&A_d, bytesA));
    CUDA_OK(hipMalloc(&B_d, bytesB));
    CUDA_OK(hipMalloc(&C_d, bytesC));

    CUDA_OK(hipMemcpy(A_d, A_h, bytesA, hipMemcpyHostToDevice));
    CUDA_OK(hipMemcpy(B_d, B_h, bytesB, hipMemcpyHostToDevice));

    dim3 block(16, 16);
    dim3 grid((N + block.x - 1) / block.x,
              (M + block.y - 1) / block.y);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    matmul_kernel<<<grid, block>>>(A_d, B_d, C_d, M, K, N);
    hipError_t launch_err = hipGetLastError();
    if (launch_err != hipSuccess) {
        std::fprintf(stderr, "Kernel launch failed: %s\n",
                     hipGetErrorString(launch_err));
        hipFree(A_d); hipFree(B_d); hipFree(C_d);
        return 3;
    }
    hipEventRecord(stop);
    CUDA_OK(hipDeviceSynchronize());

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("CUDA time: %f milliseconds\n", milliseconds);

    CUDA_OK(hipMemcpy(C_h, C_d, bytesC, hipMemcpyDeviceToHost));

    hipFree(A_d); hipFree(B_d); hipFree(C_d);
    return 0;
}
