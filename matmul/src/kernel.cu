// src/kernel.cu
#include <hip/hip_runtime.h>
#include <cstdio>

extern "C" __global__
void matmul_kernel(const float* __restrict__ A,
                   const float* __restrict__ B,
                   float* __restrict__ C,
                   int M, int K, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        float s = 0.0f;
        for (int e = 0; e < K; ++e) {
            s += A[row * K + e] * B[e * N + col];
        }
        C[row * N + col] = s;
    }
}

static inline int divUp(int a, int b) { return (a + b - 1) / b; }

// Exported C ABI for the binding TU.
extern "C"
int matmul_cuda(const float* A, const float* B, float* C, int M, int K, int N)
{
    if (!A || !B || !C || M <= 0 || K <= 0 || N <= 0) {
        std::fprintf(stderr, "matmul_cuda: invalid args\n");
        return -1;
    }

    hipError_t err = hipSuccess;
    float *dA = nullptr, *dB = nullptr, *dC = nullptr;

    // ✅ Declare launch config BEFORE any possible `goto` to avoid "bypasses initialization".
    dim3 block(16, 16);
    dim3 grid(divUp(N, block.x), divUp(M, block.y));

    size_t bytesA = (size_t)M * K * sizeof(float);
    size_t bytesB = (size_t)K * N * sizeof(float);
    size_t bytesC = (size_t)M * N * sizeof(float);

    if ((err = hipMalloc((void**)&dA, bytesA)) != hipSuccess) goto cleanup;
    if ((err = hipMalloc((void**)&dB, bytesB)) != hipSuccess) goto cleanup;
    if ((err = hipMalloc((void**)&dC, bytesC)) != hipSuccess) goto cleanup;

    if ((err = hipMemcpy(dA, A, bytesA, hipMemcpyHostToDevice)) != hipSuccess) goto cleanup;
    if ((err = hipMemcpy(dB, B, bytesB, hipMemcpyHostToDevice)) != hipSuccess) goto cleanup;

    matmul_kernel<<<grid, block>>>(dA, dB, dC, M, K, N);
    if ((err = hipGetLastError()) != hipSuccess) goto cleanup;
    if ((err = hipDeviceSynchronize()) != hipSuccess) goto cleanup;

    if ((err = hipMemcpy(C, dC, bytesC, hipMemcpyDeviceToHost)) != hipSuccess) goto cleanup;

cleanup:
    if (err != hipSuccess) {
        std::fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
    }
    if (dA) hipFree(dA);
    if (dB) hipFree(dB);
    if (dC) hipFree(dC);
    return (err == hipSuccess) ? 0 : (int)err;
}
